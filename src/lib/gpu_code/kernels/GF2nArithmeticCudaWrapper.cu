#include "hip/hip_runtime.h"
/*
 * cubffa (CUda Binary Finite Field Arithmetic library) provides 
 * functions for large binary galois field arithmetic on GPUs. 
 * Besides CUDA it is also possible to extend cubffa to any other 
 * underlying framework.
 * Copyright (C) 2016  Dominik Stamm
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
 
#include "GF2nArithmeticCudaKernels.h"
#include "GF2nArithmeticCudaWrapper.h"
#include <iostream>
#include <sys/time.h>

namespace libcumffa {
	namespace gpu {

		namespace cuda {

			uint32 g_hMaxThreadsPerBlock = 0;
			uint32 g_hSharedMemPerBlock = 0;
			uint32 g_hMultiProcessorCount = 0;
			uint32 g_hWarpSize = 0;
			uint32 g_hMaxThreadsPerMultiProcessor = 0;
			hipStream_t g_stream_0;
			hipStream_t g_stream_1;


			// return the time in milliseconds
			double cpuSecond()
			{
			    struct timeval tp;
			    gettimeofday(&tp, NULL);
			    return ((double)tp.tv_sec * 1000 + (double)tp.tv_usec * 1.e-3);
			}

			void loadPoroperties() 
			{
				cudaGetDeviceProperies(&g_hMaxThreadsPerBlock, &g_hSharedMemPerBlock, &g_hMultiProcessorCount, &g_hWarpSize, &g_hMaxThreadsPerMultiProcessor);
			}

			void device_init()
			{
				//hipDeviceReset();
				loadPoroperties();
				hipStreamCreate(&g_stream_0);
				hipStreamCreate(&g_stream_1);
			}

			/**************************************************************************\

                               sync calls

			\**************************************************************************/

			double device_allocate( CUDA_BIGNUM **d_x, uint32 num_bytes )
			{
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start);

				// allocate space for the device value
				CudaSafeCall(hipMalloc((void **)d_x, num_bytes));
				
				hipEventRecord(stop);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double device_allocate_pinned( CUDA_BIGNUM **d_x, uint32 num_bytes )
			{
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start);
				
				CudaSafeCall(hipHostAlloc((void **)d_x, num_bytes, hipHostMallocDefault));
				
				hipEventRecord(stop);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;	
			}

			double device_delete( CUDA_BIGNUM *d_x )
			{
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start);
				
				// free all device values
				CudaSafeCall(hipFree(d_x));
				
				hipEventRecord(stop);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double device_delete_pinned( CUDA_BIGNUM *d_x )
			{
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start);
				
				CudaSafeCall(hipHostFree(d_x));
				
				hipEventRecord(stop);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double device_set( CUDA_BIGNUM *d_x, CUDA_BIGNUM *h_x, uint32 num_bytes )
			{
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start);

				// copy values to the device
				CudaSafeCall(hipMemcpy(d_x, h_x, num_bytes, hipMemcpyHostToDevice));
				
				hipEventRecord(stop);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double device_swapBytes( CUDA_BIGNUM *d_x, uint32 num_chunks )
			{
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				uint32 num_threads = min(g_hMaxThreadsPerBlock, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / g_hMaxThreadsPerBlock);

				hipEventRecord(start);

				cudaSwapBytes<<<num_blocks, num_threads>>>(d_x, num_chunks);			

				hipEventRecord(stop);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double device_copy( CUDA_BIGNUM *d_y, CUDA_BIGNUM *d_x, uint32 num_bytes )	
			{
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start);

				// copy values from one device value to another
				CudaSafeCall(hipMemcpy(d_y, d_x, num_bytes, hipMemcpyDeviceToDevice));
				
				hipEventRecord(stop);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double device_get( CUDA_BIGNUM *h_x, CUDA_BIGNUM *d_x, uint32 num_bytes )
			{
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start);

				CudaSafeCall(hipMemcpy(h_x, d_x, num_bytes, hipMemcpyDeviceToHost));
				
				hipEventRecord(stop);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double device_clear( CUDA_BIGNUM *d_x, uint32 num_chunks )
			{
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				dim3 threadsPerBlock(min(g_hMaxThreadsPerBlock, (uint32)num_chunks));
				dim3 numBlocks(ceil((double)num_chunks / threadsPerBlock.x));

				hipEventRecord(start);

				// allocate space for the device value
				cudaSet0Kernel<<<numBlocks, threadsPerBlock>>>(d_x, num_chunks);
				
				hipEventRecord(stop);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			/**************************************************************************\

                               async calls

			\**************************************************************************/

			void device_set_async( CUDA_BIGNUM *d_x, CUDA_BIGNUM *h_x, uint32 num_bytes )
			{
				// copy values to the device
				CudaSafeCall(hipMemcpyAsync(d_x, h_x, num_bytes, hipMemcpyHostToDevice, g_stream_0));
			}

			void device_copy_async( CUDA_BIGNUM *d_y, CUDA_BIGNUM *d_x, uint32 num_bytes )	
			{
				// copy values from one device value to another
				CudaSafeCall(hipMemcpyAsync(d_y, d_x, num_bytes, hipMemcpyDeviceToDevice, g_stream_0));
			}

			void device_get_async( CUDA_BIGNUM *h_x, CUDA_BIGNUM *d_x, uint32 num_bytes )
			{
				CudaSafeCall(hipMemcpyAsync(h_x, d_x, num_bytes, hipMemcpyDeviceToHost, g_stream_0));
			}

			void device_hexDump( char *h_desc, void *d_addr, int len )
			{
				std::cout << h_desc << std::endl;
				std::cout << strlen(h_desc) << std::endl;
				char *d_desc;
				CudaSafeCall(hipMalloc((void **)&d_desc, strlen(h_desc) + 1));
				CudaSafeCall(hipMemcpy(d_desc, h_desc, strlen(h_desc) + 1, hipMemcpyHostToDevice));
				cudaHexDumpKernel<<<1, 1>>>(d_desc, d_addr, len);
				CudaSafeCall(hipFree(d_desc));
				hipDeviceSynchronize();
				CudaCheckError();			
			}

			/**************************************************************************\

                               Addition

			\**************************************************************************/			

			double parAdd( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(g_hMaxThreadsPerBlock, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / g_hMaxThreadsPerBlock);

				double iStart, iElaps;
				iStart = cpuSecond();

				cudaParAddKernel<<<num_blocks, num_threads>>>(x, y, res, num_chunks);
				hipDeviceSynchronize();

				iElaps = cpuSecond() - iStart;

				CudaCheckError();

				return iElaps;
			}

			double parAddLoop( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, uint32 num_threads, uint32 num_blocks, CUDA_BIGNUM *res )
			{
				double iStart, iElaps;
				iStart = cpuSecond();

				cudaParAddLoopKernel<<<num_blocks, num_threads>>>(x, y, res, num_chunks);
				hipDeviceSynchronize();

				iElaps = cpuSecond() - iStart;

				CudaCheckError();

				return iElaps;				
			}

			double parAddTime( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res, uint32 *times )
			{
				uint32 num_threads = min(g_hMaxThreadsPerBlock, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / g_hMaxThreadsPerBlock);

				// allocate memory for timer
				timing_stats *d_timer = NULL;
				CudaSafeCall(hipMalloc((void **)&d_timer, sizeof(timing_stats)));

				double iStart, iElaps;
				iStart = cpuSecond();

				cudaParAddTimeKernel<<<num_blocks, num_threads>>>(x, y, res, num_chunks, d_timer);
				hipDeviceSynchronize();

				iElaps = cpuSecond() - iStart;

				CudaCheckError();

				timing_stats h_timer;
				CudaSafeCall(hipMemcpy(&h_timer, d_timer, sizeof(timing_stats), hipMemcpyDeviceToHost));

				times[0] = (uint32)(h_timer.time2 - h_timer.time1);
				times[1] = (uint32)(h_timer.time3 - h_timer.time2);
				times[2] = (uint32)(h_timer.time4 - h_timer.time3);

				CudaSafeCall(hipFree(d_timer));

				return iElaps;
			}

			double parAddWithEvents( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(g_hMaxThreadsPerBlock, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / g_hMaxThreadsPerBlock);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start);

				cudaParAddKernel<<<num_blocks, num_threads>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double parAddOwnStream( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(g_hMaxThreadsPerBlock, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / g_hMaxThreadsPerBlock);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				hipEventRecord(start, g_stream_0);

				cudaParAddKernel<<<num_blocks, num_threads, 0, g_stream_0>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop, g_stream_0);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double parAddOwnStream1024Threads( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(1024, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / 1024);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				hipEventRecord(start, g_stream_0);

				cudaParAddKernel<<<num_blocks, num_threads, 0, g_stream_0>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop, g_stream_0);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double parAddOwnStream512Threads( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(512, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / 512);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				hipEventRecord(start, g_stream_0);

				cudaParAddKernel<<<num_blocks, num_threads, 0, g_stream_0>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop, g_stream_0);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double parAddOwnStream256Threads( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(256, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / 256);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				hipEventRecord(start, g_stream_0);

				cudaParAddKernel<<<num_blocks, num_threads, 0, g_stream_0>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop, g_stream_0);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}			

			double parAddOwnStream128Threads( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(128, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / 128);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				hipEventRecord(start, g_stream_0);

				cudaParAddKernel<<<num_blocks, num_threads, 0, g_stream_0>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop, g_stream_0);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}					

			double parAddMultiStreams( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(g_hMaxThreadsPerBlock, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / g_hMaxThreadsPerBlock);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				hipEventRecord(start, g_stream_0);

				cudaParAddKernel<<<num_blocks, num_threads, 0, g_stream_0>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop, g_stream_0);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double parAdd2OwnStream( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(g_hMaxThreadsPerBlock, (uint32)ceil((double)num_chunks / 2));
				uint32 num_blocks = ceil((double)num_chunks / g_hMaxThreadsPerBlock);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				hipEventRecord(start, g_stream_0);

				cuda2ParAddKernel<<<num_blocks, num_threads, 0, g_stream_0>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop, g_stream_0);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double parAdd4OwnStream( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(g_hMaxThreadsPerBlock, (uint32)ceil((double)num_chunks / 4));
				uint32 num_blocks = ceil((double)num_chunks / g_hMaxThreadsPerBlock);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				hipEventRecord(start, g_stream_0);

				cuda4ParAddKernel<<<num_blocks, num_threads, 0, g_stream_0>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop, g_stream_0);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			double parAdd8OwnStream( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(g_hMaxThreadsPerBlock, (uint32)ceil((double)num_chunks / 8));
				uint32 num_blocks = ceil((double)num_chunks / g_hMaxThreadsPerBlock);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				hipEventRecord(start, g_stream_0);

				cuda8ParAddKernel<<<num_blocks, num_threads, 0, g_stream_0>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop, g_stream_0);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}	

			double parAddSharedMem( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM *res )
			{
				uint32 num_threads = min(g_hMaxThreadsPerBlock, (uint32)num_chunks);
				uint32 num_blocks = ceil((double)num_chunks / g_hMaxThreadsPerBlock);

				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);

				cudaParAddKernel<<<1, 1>>>(x, y, res, num_chunks);

				hipEventRecord(start, g_stream_0);

				cudaParAddKernel<<<num_blocks, num_threads, num_chunks*num_threads, g_stream_0>>>(x, y, res, num_chunks);
				
				hipEventRecord(stop, g_stream_0);

				hipEventSynchronize(stop);
				
				float iElaps;
				hipEventElapsedTime(&iElaps, start, stop);

				CudaCheckError();

				return (double)iElaps;
			}

			/**************************************************************************\

                               Multiplication

			\**************************************************************************/			

			double parMul( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM* irred_poly, uint32 indx_mask_bit, CUDA_BIGNUM *res )
			{
				device_clear(res, num_chunks);

				double iStart, iElaps;
				iStart = cpuSecond();

				cudaParMulKernel<<<1, 1>>>(x, y, res, num_chunks, irred_poly, indx_mask_bit);	
				hipDeviceSynchronize();

				iElaps = cpuSecond() - iStart;

				CudaCheckError();

				return iElaps;
			}

			void parRedImpl( CUDA_BIGNUM *a, uint32 numChunksA, CUDA_BIGNUM maskRed, CUDA_BIGNUM *res, uint32 numChunksRes)
			{
				// res(x) = a(x) % x^n -> only take n bits of the right side
				// First copy all chunks that remain from a to res
				device_copy(res, &a[numChunksA - numChunksRes], numChunksRes * CUDA_BIGNUM_SIZE_BYTES);
				
				// Then ajust the first chunk of res with the mask
				cudaMaskFirstChunkKernel<<<1, 1>>>(res, maskRed);
			}

			void parShiftLeftImpl( CUDA_BIGNUM *x, uint32 numChunks, uint32 numBitsToShift, CUDA_BIGNUM *res, uint32 numChunksRes )
			{
				dim3 threadsPerBlock(min(g_hMaxThreadsPerBlock, numChunksRes));
				dim3 numBlocks(ceil((double)numChunksRes / threadsPerBlock.x));

				uint32 numBitsToShiftInBlock = numBitsToShift % CUDA_BIGNUM_SIZE_BITS;
				uint32 blockOffset = (uint32)(numBitsToShift / CUDA_BIGNUM_SIZE_BITS);

				CUDA_BIGNUM maskRight = ~((CUDA_BIGNUM)0);
				if( CUDA_BIGNUM_SIZE_BITS - numBitsToShiftInBlock < CUDA_BIGNUM_SIZE_BITS ) 
					maskRight -= ((CUDA_BIGNUM)pow((double)2, (double)CUDA_BIGNUM_SIZE_BITS - numBitsToShiftInBlock) - 1);
				else
					maskRight = 0;
				CUDA_BIGNUM maskLeft = ~((CUDA_BIGNUM)0) - maskRight;

				cudaShiftLeftKernel<<<numBlocks, threadsPerBlock>>>(x, numChunks, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, res, numChunksRes);
			}

			void parShiftRightImpl( CUDA_BIGNUM *x, uint32 numChunks, uint32 numBitsToShift, CUDA_BIGNUM *res, uint32 numChunksRes )
			{
				dim3 threadsPerBlock(min(g_hMaxThreadsPerBlock, numChunksRes));
				dim3 numBlocks(ceil((double)numChunksRes / threadsPerBlock.x));

				uint32 numBitsToShiftInBlock = numBitsToShift % CUDA_BIGNUM_SIZE_BITS;
				uint32 blockOffset = (uint32)(numBitsToShift / CUDA_BIGNUM_SIZE_BITS);
				CUDA_BIGNUM maskLeft  =  (CUDA_BIGNUM)pow((double)2, (double)(numBitsToShiftInBlock)) - 1;
				CUDA_BIGNUM maskRight =  ~((CUDA_BIGNUM)0) - maskLeft;

				cudaShiftRightKernel<<<numBlocks, threadsPerBlock>>>(x, numChunks, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, res, numChunksRes);
			}

			void parGetElemDegreeImpl( CUDA_BIGNUM *x, uint32 numChunks, CUDA_BIGNUM *foundChunkIndex, CUDA_BIGNUM *elemDegree )
			{
				dim3 threadsPerBlock(min(g_hMaxThreadsPerBlock, numChunks));
				dim3 numBlocks(ceil((double)numChunks / threadsPerBlock.x));

				cudaGetFirstNonEmptyChunkIndexKernel<<<numBlocks, threadsPerBlock>>>(x, numChunks, foundChunkIndex);
				cudaCalcElemDegreeKernel<<<1, 1>>>(x, numChunks, foundChunkIndex, elemDegree);
			}

			void parMulChunkedImpl( CUDA_BIGNUM *x, CUDA_BIGNUM *y, CUDA_BIGNUM *tmpArr, uint32 numChunks, CUDA_BIGNUM *res )
			{
				device_clear(tmpArr, 4 * numChunks * numChunks);

				dim3 threadsPerBlock(4, 16);
				dim3 numBlocks(ceil((double)numChunks / threadsPerBlock.x), ceil((double)2 * numChunks / threadsPerBlock.y));

				cudaCreateChunkProdArrayKernel<<<numBlocks, threadsPerBlock>>>(x, y, tmpArr, numChunks);	

				dim3 threadsPerBlock2(64, 1); 
				dim3 numBlocks2(ceil((double)numChunks / threadsPerBlock2.x), 
					ceil((double)2 * numChunks / threadsPerBlock2.y));

				cudaParDiagBlockSumKernel<<<numBlocks2, threadsPerBlock2>>>(tmpArr, numChunks);

				dim3 threadsPerBlock3(min(g_hMaxThreadsPerBlock, (uint32)2 * numChunks));
				dim3 numBlocks3(ceil((double)2 * numChunks / threadsPerBlock3.x));

				cudaParChunkSumKernel<<<numBlocks3, threadsPerBlock3>>>(tmpArr, numChunks, numBlocks2.x, 2 * threadsPerBlock2.x, res);

				hipDeviceSynchronize();

				CudaCheckError();
			}

			/*
				Q1(x) = A(x) / x^n -> Shift um n Bits nach rechts
				Q2(x) = M(x) * Q1(x) -> Erg der Größe 2n
				Q3(x) = Q2(x) / x^n -> Shift um n Bits nach rechts
				R1(x) = A(x) % x^n -> nur rechten n Bits
				R2(x) = M(x) * Q3(x) % x^n
				R(x) = R1(x) + R2(x)	
			*/
			void parBarRedImpl( CUDA_BIGNUM *a, uint32 numChunksA, CUDA_BIGNUM *tmpArr, CUDA_BIGNUM *tmp,
				CUDA_BIGNUM *q1, CUDA_BIGNUM *q2, CUDA_BIGNUM *q3, CUDA_BIGNUM *r1, CUDA_BIGNUM *r2, 
				CUDA_BIGNUM *irredPoly, uint32 fieldSize, 
				uint32 numBitsToShiftInBlock, uint32 blockOffset,
				CUDA_BIGNUM maskLeft, CUDA_BIGNUM maskRight,
				CUDA_BIGNUM *res, uint32 numChunksRes )
			{	
				dim3 threadsPerBlock(min(g_hMaxThreadsPerBlock, numChunksRes));
				dim3 numBlocks(ceil((double)numChunksRes / threadsPerBlock.x));

				// Q1(x) = A(x) / x^n -> shift n bits right
				parShiftRightImpl(a, numChunksA, fieldSize, q1, numChunksRes);
				//cudaShiftRightKernel<<<numBlocks, threadsPerBlock>>>(a, numChunksA, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, q1, numChunksRes);

				// Q2(x) = M(x) * Q1(x) -> q2 has 2n bits
				parMulChunkedImpl(q1, irredPoly, tmpArr, numChunksRes, q2);

				// Q3(x) = Q2(x) / x^n -> shift n bits left
				parShiftRightImpl(q2, numChunksA, fieldSize, q3, numChunksRes);
				//cudaShiftRightKernel<<<numBlocks, threadsPerBlock>>>(q2, numChunksA, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, q3, numChunksRes);

				// R1(x) = A(x) % x^n -> take n lsb bits
				parRedImpl(a, numChunksA, maskLeft, r1, numChunksRes);

				// R2(x) = M(x) * Q3(x) % x^n
				parMulChunkedImpl(q3, irredPoly, tmpArr, numChunksRes, tmp);
				parRedImpl(tmp, numChunksA, maskLeft, r2, numChunksRes);

				// R(x) = R1(x) + R2(x)
				cudaParAddKernel<<<numBlocks, threadsPerBlock>>>(r1, r2, res, numChunksRes );

				hipDeviceSynchronize();

				CudaCheckError();
			}

			void parInverseElementImpl( CUDA_BIGNUM *x, uint32 numChunks, 
				CUDA_BIGNUM *irredPoly, CUDA_BIGNUM *tmp,
				CUDA_BIGNUM *s, CUDA_BIGNUM *r, CUDA_BIGNUM *v, CUDA_BIGNUM *u,
				CUDA_BIGNUM *degR, CUDA_BIGNUM *degS, CUDA_BIGNUM *foundChunkIndex, 
				CUDA_BIGNUM *res )
			{
				dim3 threadsPerBlock(min(g_hMaxThreadsPerBlock, numChunks));
				dim3 numBlocks(ceil((double)numChunks / threadsPerBlock.x));

				// S(x) = G(x)
				device_copy(s, irredPoly, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				// R(X) = A(x)
				device_copy(r, x, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				// V(x) = 0
				device_clear(v, numChunks);
				// U(x) = 1
				cudaSet1Kernel<<<numBlocks, threadsPerBlock>>>(u, numChunks);

				CUDA_BIGNUM h_degR, h_degS, h_degDelta;

				// degR = deg(R(x))
				parGetElemDegreeImpl(r, numChunks, foundChunkIndex, degR);
				device_get(&h_degR, degR, CUDA_BIGNUM_SIZE_BYTES);

				while( h_degR )
				{
					parGetElemDegreeImpl(s, numChunks, foundChunkIndex, degS);
					device_get(&h_degS, degS, CUDA_BIGNUM_SIZE_BYTES);

					if( h_degR > h_degS )
					{
						// printf("hit swap\n");
						// swap pointers
						CUDA_BIGNUM *ptrTmp = NULL;
						// temp := S(x); S(x) := R(x); R(x) := temp;
						ptrTmp = s; s = r, r = ptrTmp;
						// temp := V (x); V (x) := U(x); U(x) := temp;
						ptrTmp = v; v = u; u = ptrTmp;
						
						h_degDelta = h_degR - h_degS;
					}
					else
					{
						h_degDelta = h_degS - h_degR;
					}

					// S(x) = S(x) − x^degDelta * R(x)
					device_clear(tmp, numChunks);
					parShiftLeftImpl(r, numChunks, h_degDelta, tmp, numChunks);
					cudaParAddKernel<<<numBlocks, threadsPerBlock>>>(s, tmp, numChunks);

					// V(x) = V(x) - x^degDelta * U(x)
					device_clear(tmp, numChunks);
					parShiftLeftImpl(u, numChunks, h_degDelta, tmp, numChunks);
					cudaParAddKernel<<<numBlocks, threadsPerBlock>>>(v, tmp, numChunks);

					// degR = deg(R(x))
					parGetElemDegreeImpl(r, numChunks, foundChunkIndex, degR);
					device_get(&h_degR, degR, CUDA_BIGNUM_SIZE_BYTES);
				}

				hipDeviceSynchronize();

				// return U(x) as the result
				device_copy(res, u, CUDA_BIGNUM_SIZE_BYTES * numChunks);

				CudaCheckError();
			}

			double parMulChunkedBarRed( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 numChunks, CUDA_BIGNUM *irredPoly, uint32 fieldSize, CUDA_BIGNUM *res )
			{
				double iStart, iElaps;

				uint32 matrixWidth = 2 * numChunks;
				uint32 matrixHeight = 2 * numChunks;

				device_clear(res, numChunks);

				// allocate result array
				CUDA_BIGNUM *arr = NULL;
				device_allocate(&arr, CUDA_BIGNUM_SIZE_BYTES * matrixWidth * matrixHeight);

				// allocate space for tmp result befor reduction
				// of size 2 * num_chunks
				CUDA_BIGNUM *tmp, *resTmp = NULL;
				device_allocate(&resTmp, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);
				device_allocate(&tmp, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);

				CUDA_BIGNUM *q1, *q2, *q3, *r1, *r2 = NULL;
				device_allocate(&q1, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&q2, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);
				device_allocate(&q3, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&r1, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&r2, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);

				uint32 numBitsToShiftInBlock = fieldSize % CUDA_BIGNUM_SIZE_BITS;
				uint32 blockOffset = (uint32)(fieldSize / CUDA_BIGNUM_SIZE_BITS);
				CUDA_BIGNUM maskLeft  =  (CUDA_BIGNUM)pow((double)2, (double)(numBitsToShiftInBlock)) - 1;
				CUDA_BIGNUM maskRight =  ~((CUDA_BIGNUM)0) - maskLeft;

				// {
				// 	uint32 *foundChunkIndex, *elemDegree;
				// 	device_allocate(&foundChunkIndex, 4);
				// 	device_allocate(&elemDegree, 4);

				// 	uint32 h_elemDegree;

				// 	parGetElemDegreeImpl(x, numChunks, foundChunkIndex, elemDegree);
				// 	device_get(&h_elemDegree, elemDegree, 4);

				// 	std::cout << "element has degree " << h_elemDegree << std::endl;

				// 	device_delete(elemDegree);
				// 	device_delete(foundChunkIndex);
				// }
				
				//Test Left Shift
				{
					CUDA_BIGNUM *tmp_x = NULL;
					device_allocate(&tmp_x, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);

					parShiftLeftImpl(x, numChunks, fieldSize, tmp_x, 2 * numChunks);
					parShiftRightImpl(tmp_x, 2 * numChunks, fieldSize, x, numChunks);

					hipDeviceSynchronize();
					
					device_delete(tmp_x);
				}

				iStart = cpuSecond();

				parMulChunkedImpl(x, y, arr, numChunks, resTmp);
				parBarRedImpl(resTmp, matrixHeight, arr, tmp, q1, q2, q3, r1, r2, irredPoly, fieldSize, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, res, numChunks);

				iElaps = cpuSecond() - iStart;

				device_delete(q1);
				device_delete(q2);
				device_delete(q3);
				device_delete(r1);
				device_delete(r2);

				device_delete(tmp);
				device_delete(resTmp);
				device_delete(arr);

				return iElaps;				
			}		

			/**************************************************************************\

                               Exponentiation

			\**************************************************************************/

            double parExponentiation( CUDA_BIGNUM *x, uint32 k, uint32 numChunks, CUDA_BIGNUM *irredPoly, uint32 fieldSize, CUDA_BIGNUM *res )
            {
				double iStart, iElaps;

				uint32 matrixWidth = 2 * numChunks;
				uint32 matrixHeight = 2 * numChunks;

				device_clear(res, CUDA_BIGNUM_SIZE_BYTES * numChunks);

				// allocate result array
				CUDA_BIGNUM *arr = NULL;
				device_allocate(&arr, CUDA_BIGNUM_SIZE_BYTES * matrixWidth * matrixHeight);

				// allocate space for tmp result befor reduction
				// of size 2 * num_chunks
				CUDA_BIGNUM *tmp, *resTmp = NULL;
				device_allocate(&resTmp, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);
				device_allocate(&tmp, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);

				CUDA_BIGNUM *q1, *q2, *q3, *r1, *r2 = NULL;
				device_allocate(&q1, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&q2, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);
				device_allocate(&q3, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&r1, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&r2, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);

				uint32 numBitsToShiftInBlock = fieldSize % CUDA_BIGNUM_SIZE_BITS;
				uint32 blockOffset = (uint32)(fieldSize / CUDA_BIGNUM_SIZE_BITS);
				CUDA_BIGNUM maskLeft  =  (CUDA_BIGNUM)pow((double)2, (double)(numBitsToShiftInBlock)) - 1;
				CUDA_BIGNUM maskRight =  ~((CUDA_BIGNUM)0) - maskLeft;

				iStart = cpuSecond();

				if( !k )
				{
					dim3 threadsPerBlock(min(g_hMaxThreadsPerBlock, numChunks));
					dim3 numBlocks(ceil((double)numChunks / threadsPerBlock.x));
					cudaSet1Kernel<<<numBlocks, threadsPerBlock>>>(res, numChunks);
				}
				else
				{
					// Calculate the mask 
				    uint32 h = ~(~0u >> 1); 
	    			while( !(k & h) ) 
	    				h >>= 1;

	    			// initial copy x to res
	    			device_copy(res, x, CUDA_BIGNUM_SIZE_BYTES * numChunks);

	    			// As long as the mask isn't 0 -> do exponentiation
				    while( h >>= 1 )
				    {
				    	// Calculate res = res^2
				    	parMulChunkedImpl(res, res, arr, numChunks, resTmp);
						parBarRedImpl(resTmp, matrixHeight, arr, tmp, q1, q2, q3, r1, r2, irredPoly, fieldSize, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, res, numChunks);

						// If bit is set -> calculate res = res * x
				        if( k & h )
				        {
				        	parMulChunkedImpl(res, x, arr, numChunks, resTmp);
							parBarRedImpl(resTmp, matrixHeight, arr, tmp, q1, q2, q3, r1, r2, irredPoly, fieldSize, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, res, numChunks);
				        }
				    }
				}

				iElaps = cpuSecond() - iStart;

				device_delete(q1);
				device_delete(q2);
				device_delete(q3);
				device_delete(r1);
				device_delete(r2);

				device_delete(tmp);
				device_delete(resTmp);
				device_delete(arr);

				return iElaps;	
            }

			/**************************************************************************\

                               Inverse

			\**************************************************************************/

			double parInverseElement( CUDA_BIGNUM *x, uint32 numChunks, CUDA_BIGNUM *irredPoly, CUDA_BIGNUM *res )
			{
				double iStart, iElaps;

				CUDA_BIGNUM *tmp, *s, *r, *v, *u;
				device_allocate(&tmp, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&s, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&r, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&v, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&u, CUDA_BIGNUM_SIZE_BYTES * numChunks);

				CUDA_BIGNUM *degR, *degS, *foundChunkIndex;
				device_allocate(&degR, CUDA_BIGNUM_SIZE_BYTES);
				device_allocate(&degS, CUDA_BIGNUM_SIZE_BYTES);
				device_allocate(&foundChunkIndex, CUDA_BIGNUM_SIZE_BYTES);

				iStart = cpuSecond();

				parInverseElementImpl(x, numChunks, irredPoly, tmp, s, r, v, u, degR, degS, foundChunkIndex, res);

				iElaps = cpuSecond() - iStart;

				device_delete(foundChunkIndex);
				device_delete(degS);
				device_delete(degR);
				
				device_delete(u);
				device_delete(v);
				device_delete(r);
				device_delete(s);
				device_delete(tmp);

				return iElaps;
			}

			double parInverseElementWithExp( CUDA_BIGNUM *x, uint32 numChunks, CUDA_BIGNUM *irredPoly, uint32 fieldSize, CUDA_BIGNUM *res )
            {
				double iStart, iElaps;

				uint32 matrixWidth = 2 * numChunks;
				uint32 matrixHeight = 2 * numChunks;

				device_clear(res, CUDA_BIGNUM_SIZE_BYTES * numChunks);

				// allocate result array
				CUDA_BIGNUM *arr = NULL;
				device_allocate(&arr, CUDA_BIGNUM_SIZE_BYTES * matrixWidth * matrixHeight);

				// allocate space for tmp result befor reduction
				// of size 2 * num_chunks
				CUDA_BIGNUM *tmp, *resTmp = NULL;
				device_allocate(&resTmp, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);
				device_allocate(&tmp, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);

				CUDA_BIGNUM *q1, *q2, *q3, *r1, *r2 = NULL;
				device_allocate(&q1, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&q2, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);
				device_allocate(&q3, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&r1, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&r2, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);

				uint32 numBitsToShiftInBlock = fieldSize % CUDA_BIGNUM_SIZE_BITS;
				uint32 blockOffset = (uint32)(fieldSize / CUDA_BIGNUM_SIZE_BITS);
				CUDA_BIGNUM maskLeft  =  (CUDA_BIGNUM)pow((double)2, (double)(numBitsToShiftInBlock)) - 1;
				CUDA_BIGNUM maskRight =  ~((CUDA_BIGNUM)0) - maskLeft;

				iStart = cpuSecond();

    			// initial copy x to res
    			device_copy(res, x, CUDA_BIGNUM_SIZE_BYTES * numChunks);

    			// Mask is always (2^fieldSize)-2
			    for( int i=0; i<fieldSize - 2; ++i )
			    {
			    	// Calculate res = res^2
			    	parMulChunkedImpl(res, res, arr, numChunks, resTmp);
					parBarRedImpl(resTmp, matrixHeight, arr, tmp, q1, q2, q3, r1, r2, irredPoly, fieldSize, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, res, numChunks);

					// Calculate res = res * x
			        parMulChunkedImpl(res, x, arr, numChunks, resTmp);
					parBarRedImpl(resTmp, matrixHeight, arr, tmp, q1, q2, q3, r1, r2, irredPoly, fieldSize, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, res, numChunks);
			    }

			    // Calculate res = res^2
			    parMulChunkedImpl(res, res, arr, numChunks, resTmp);
				parBarRedImpl(resTmp, matrixHeight, arr, tmp, q1, q2, q3, r1, r2, irredPoly, fieldSize, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, res, numChunks);

				iElaps = cpuSecond() - iStart;

				device_delete(q1);
				device_delete(q2);
				device_delete(q3);
				device_delete(r1);
				device_delete(r2);

				device_delete(tmp);
				device_delete(resTmp);
				device_delete(arr);

				return iElaps;	
            }

            /**************************************************************************\

                               Polynomial Evaluation

			\**************************************************************************/

            double parEvaluatePoly( CUDA_BIGNUM *coeffs, uint32 numCoeffs, CUDA_BIGNUM *x, uint32 numChunks, CUDA_BIGNUM *irredPoly, uint32 fieldSize, CUDA_BIGNUM *res )
            {
				double iStart, iElaps;

				uint32 matrixWidth = 2 * numChunks;
				uint32 matrixHeight = 2 * numChunks;

				device_clear(res, CUDA_BIGNUM_SIZE_BYTES * numChunks);

				// allocate result array
				CUDA_BIGNUM *arr = NULL;
				device_allocate(&arr, CUDA_BIGNUM_SIZE_BYTES * matrixWidth * matrixHeight);

				// allocate space for tmp result befor reduction
				// of size 2 * num_chunks
				CUDA_BIGNUM *tmp, *resTmp, *xTmp, *resTmp2 = NULL;
				device_allocate(&resTmp, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);
				device_allocate(&tmp, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);
				device_allocate(&xTmp, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&resTmp2, CUDA_BIGNUM_SIZE_BYTES * numChunks);

				CUDA_BIGNUM *q1, *q2, *q3, *r1, *r2 = NULL;
				device_allocate(&q1, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&q2, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);
				device_allocate(&q3, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&r1, CUDA_BIGNUM_SIZE_BYTES * numChunks);
				device_allocate(&r2, CUDA_BIGNUM_SIZE_BYTES * 2 * numChunks);

				uint32 numBitsToShiftInBlock = fieldSize % CUDA_BIGNUM_SIZE_BITS;
				uint32 blockOffset = (uint32)(fieldSize / CUDA_BIGNUM_SIZE_BITS);
				CUDA_BIGNUM maskLeft  =  (CUDA_BIGNUM)pow((double)2, (double)(numBitsToShiftInBlock)) - 1;
				CUDA_BIGNUM maskRight =  ~((CUDA_BIGNUM)0) - maskLeft;

				iStart = cpuSecond();

    			// initial xTmp = x^0
    			dim3 threadsPerBlock(min(g_hMaxThreadsPerBlock, numChunks));
				dim3 numBlocks(ceil((double)numChunks / threadsPerBlock.x));

				// Initialize xTmp with 1
				cudaSet1Kernel<<<numBlocks, threadsPerBlock>>>(xTmp, numChunks);

				// Set res to last coeff
				device_copy(res, &coeffs[(numCoeffs - 1) * numChunks], CUDA_BIGNUM_SIZE_BYTES * numChunks);

    			// Iterate over all coeffs
			    for( int i=numCoeffs - 2; i>=0; --i )
			    {
			    	// Calculate xTmp = xTmp * x
					parMulChunkedImpl(xTmp, x, arr, numChunks, resTmp);
					parBarRedImpl(resTmp, matrixHeight, arr, tmp, q1, q2, q3, r1, r2, irredPoly, fieldSize, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, xTmp, numChunks);

			    	// Calculate resTmp2 = coeffs[i] * xTmp
			    	parMulChunkedImpl(&coeffs[i * numChunks], xTmp, arr, numChunks, resTmp);
					parBarRedImpl(resTmp, matrixHeight, arr, tmp, q1, q2, q3, r1, r2, irredPoly, fieldSize, numBitsToShiftInBlock, blockOffset, maskLeft, maskRight, resTmp2, numChunks);

					// Calculate res = res + resTmp2
					cudaParAddKernel<<<numBlocks, threadsPerBlock>>>(res, resTmp2, numChunks);
			    }

				iElaps = cpuSecond() - iStart;

				device_delete(q1);
				device_delete(q2);
				device_delete(q3);
				device_delete(r1);
				device_delete(r2);

				device_delete(resTmp2);
				device_delete(xTmp);
				device_delete(tmp);
				device_delete(resTmp);
				device_delete(arr);

				return iElaps;	
            }

			/**************************************************************************\

                               Helper

			\**************************************************************************/

			double createChunkProdArray( CUDA_BIGNUM *x, CUDA_BIGNUM *y, uint32 num_chunks, CUDA_BIGNUM **res )	
			{
				double iStart, iElaps;
				iStart = cpuSecond();

				cudaCreateChunkProdArrayKernel<<<1, 1>>>(x, y, *res, num_chunks);
				hipDeviceSynchronize();

				iElaps = cpuSecond() - iStart;

				CudaCheckError();

				return iElaps;
			}

			double measureKernelLaunchOverhead()
			{
				double iStart, iElaps;
				iStart = cpuSecond();

				EmptyKernel<<<1, 1>>>();	
				hipDeviceSynchronize();

				iElaps = cpuSecond() - iStart;

				CudaCheckError();

				return iElaps;	
			}


			void print( CUDA_BIGNUM *x, uint32 num_chunks )
			{
				cudaPrintKernel<<<1, num_chunks>>>(x, num_chunks);
				CudaCheckError();
			}

		}
	}
}
