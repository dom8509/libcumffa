#include "hip/hip_runtime.h"
/*
 * cubffa (CUda Binary Finite Field Arithmetic library) provides 
 * functions for large binary galois field arithmetic on GPUs. 
 * Besides CUDA it is also possible to extend cubffa to any other 
 * underlying framework.
 * Copyright (C) 2016  Dominik Stamm
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "GF2nArithmeticCudaKernels.h"

#define SIZE_CHUNK 64

////////////////////////////////////////////////////////////////////////////////
//
//  Device Contants
//
////////////////////////////////////////////////////////////////////////////////
__constant__ uint32 dMaxThreadsPerBlock;
__constant__ uint32 dSharedMemPerBlock;
__constant__ uint32 dMultiProcessorCount;
__constant__ uint32 dWarpSize;
__constant__ uint32 dMaxThreadsPerMultiProcessor;

////////////////////////////////////////////////////////////////////////////////
//
//	Host functions
//
////////////////////////////////////////////////////////////////////////////////
__host__ void cudaGetDeviceProperies( 
	uint32 *hMaxThreadsPerBlock, 
	uint32 *hSharedMemPerBlock, 
	uint32 *hMultiProcessorCount, 
	uint32 *hWarpSize, 
	uint32 *hMaxThreadsPerMultiProcessor
	)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dMaxThreadsPerBlock), (const char *)&deviceProp.maxThreadsPerBlock, sizeof(uint32), 0, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dSharedMemPerBlock), &deviceProp.sharedMemPerBlock, sizeof(uint32)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dMultiProcessorCount), &deviceProp.multiProcessorCount, sizeof(uint32)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dWarpSize), &deviceProp.warpSize, sizeof(uint32)));
	CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dMaxThreadsPerMultiProcessor), &deviceProp.maxThreadsPerMultiProcessor, sizeof(uint32)));

	*hMaxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
	*hSharedMemPerBlock = deviceProp.sharedMemPerBlock;
	*hMultiProcessorCount = deviceProp.multiProcessorCount;
	*hWarpSize = deviceProp.warpSize;
	*hMaxThreadsPerMultiProcessor = deviceProp.maxThreadsPerMultiProcessor;
}

__global__ void EmptyKernel() { }

/**************************************************************************\

                                   Addition

\**************************************************************************/

////////////////////////////////////////////////////////////////////////////////
//
//	Adds a and b and stores the result in res
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaParAddKernel( CUDA_BIGNUM *a, CUDA_BIGNUM *b, CUDA_BIGNUM *res, uint32 num_chunks ) 
{
	int32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < num_chunks )
		res[thid] = a[thid] ^ b[thid];
}

__global__ void cudaParAddKernel( CUDA_BIGNUM *a, CUDA_BIGNUM *b, uint32 num_chunks ) 
{
	int32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < num_chunks )
		a[thid] = a[thid] ^ b[thid];
}

__global__ void cudaParAddLoopKernel( CUDA_BIGNUM *a, CUDA_BIGNUM *b, CUDA_BIGNUM *res, uint32 num_chunks )
{
    for( int32 thid = blockIdx.x * blockDim.x + threadIdx.x; 
         thid < num_chunks; 
         thid += blockDim.x * gridDim.x) 
      {
          res[thid] = a[thid] ^ b[thid];
      }
}

__global__ void cudaParAddTimeKernel( CUDA_BIGNUM *a, CUDA_BIGNUM *b, CUDA_BIGNUM *res, uint32 num_chunks, timing_stats *time )
{
#ifdef LINUXINTEL32
	int32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < num_chunks )
	{
		// define registers
		asm(".reg .s32 %t<14>;\n\t"
			"mov.u32 %t0, %clock;");
        
        // load a and b to registers
        asm("cvta.to.global.u32 %%t4, %0;\n\t"
            "shl.b32 %%t5, %%r1, 2;\n\t"
            "add.s32 %%t6, %%t5, %%t4;\n\t"
            "cvta.to.global.u32 %%t7, %1;\n\t"
            "add.s32 %%t8, %%t7, %%t5;\n\t"
            "ld.global.u32 %%t9, [%%t8];\n\t"
            "ld.global.u32 %%t10, [%%t6];\n\t"
            "mov.u32 %%t1, %%clock;"
            :: "r"(a), "r"(b));

        asm("xor.b32 %t11, %t9, %t10;\n\t"
        	"mov.u32 %t2, %clock;");

        asm("cvta.to.global.u32 %%t12, %0;\n\t"
            "add.s32 %%t13, %%t12, %%t5;\n\t"
            "st.global.u32 [%%t13], %%t11;\n\t"
            "mov.u32 %%t3, %%clock;"
            :: "r"(res));

        asm volatile("mov.u32 %0, %%t0;" : "=r"(time[thid].time1) :: "memory");
        asm volatile("mov.u32 %0, %%t1;" : "=r"(time[thid].time2) :: "memory");
        asm volatile("mov.u32 %0, %%t2;" : "=r"(time[thid].time3) :: "memory");
        asm volatile("mov.u32 %0, %%t3;" : "=r"(time[thid].time4) :: "memory");
	}
#endif
}

__global__ void cuda2ParAddKernel( CUDA_BIGNUM *a, CUDA_BIGNUM *b, CUDA_BIGNUM *res, uint32 num_chunks )
{
	int32 thid = (blockIdx.x * blockDim.x * 2) + threadIdx.x;

	if( thid + blockDim.x < num_chunks )
	{
		res[thid + 0 * blockDim.x] = a[thid + 0 * blockDim.x] ^ b[thid + 0 * blockDim.x];
		res[thid + 1 * blockDim.x] = a[thid + 1 * blockDim.x] ^ b[thid + 1 * blockDim.x];
	}
}

__global__ void cuda4ParAddKernel( CUDA_BIGNUM *a, CUDA_BIGNUM *b, CUDA_BIGNUM *res, uint32 num_chunks )
{
	int32 thid = (blockIdx.x * blockDim.x * 4) + threadIdx.x;

	if( thid + 3 * blockDim.x < num_chunks )
	{
		res[thid + 0 * blockDim.x] = a[thid + 0 * blockDim.x] ^ b[thid + 0 * blockDim.x];
		res[thid + 1 * blockDim.x] = a[thid + 1 * blockDim.x] ^ b[thid + 1 * blockDim.x];
		res[thid + 2 * blockDim.x] = a[thid + 2 * blockDim.x] ^ b[thid + 2 * blockDim.x];
		res[thid + 3 * blockDim.x] = a[thid + 3 * blockDim.x] ^ b[thid + 3 * blockDim.x];
	}
}

__global__ void cuda8ParAddKernel( CUDA_BIGNUM *a, CUDA_BIGNUM *b, CUDA_BIGNUM *res, uint32 num_chunks )
{
	int32 thid = (blockIdx.x * blockDim.x * 8) + threadIdx.x;

	if( thid + 7 * blockDim.x < num_chunks )
	{
		res[thid + 0 * blockDim.x] = a[thid + 0 * blockDim.x] ^ b[thid + 0 * blockDim.x];
		res[thid + 1 * blockDim.x] = a[thid + 1 * blockDim.x] ^ b[thid + 1 * blockDim.x];
		res[thid + 2 * blockDim.x] = a[thid + 2 * blockDim.x] ^ b[thid + 2 * blockDim.x];
		res[thid + 3 * blockDim.x] = a[thid + 3 * blockDim.x] ^ b[thid + 3 * blockDim.x];
		res[thid + 4 * blockDim.x] = a[thid + 4 * blockDim.x] ^ b[thid + 4 * blockDim.x];
		res[thid + 5 * blockDim.x] = a[thid + 5 * blockDim.x] ^ b[thid + 5 * blockDim.x];
		res[thid + 6 * blockDim.x] = a[thid + 6 * blockDim.x] ^ b[thid + 6 * blockDim.x];
		res[thid + 7 * blockDim.x] = a[thid + 7 * blockDim.x] ^ b[thid + 7 * blockDim.x];
	}
}

__global__ void cudaParAddSharedMemKernel( CUDA_BIGNUM *a, CUDA_BIGNUM *b, CUDA_BIGNUM *res, uint32 num_chunks ) 
{
	extern __shared__ CUDA_BIGNUM sa[];
	extern __shared__ CUDA_BIGNUM sb[];
	extern __shared__ CUDA_BIGNUM sr[];

	int32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < num_chunks )
		res[thid] = a[thid] ^ b[thid];
}

/**************************************************************************\

                               Multiplication

\**************************************************************************/

__global__ void cudaParMulKernel( CUDA_BIGNUM *a, CUDA_BIGNUM *b, CUDA_BIGNUM *res, uint32 num_chunks, CUDA_BIGNUM *irred_poly, uint32 indx_mask_bit ) 
{
	for( uint32 i=num_chunks; i>0; --i ) 
	{
		for( uint32 j=0; j<CUDA_BIGNUM_SIZE_BITS; ++j ) 
		{
			if( isbitset(b[i-1], j) ) 
			{
				for( uint32 k=0; k<num_chunks; ++k )
				{
					res[k] ^= a[k];
				}
			} 
			
			cudaBitShiftLeft1(a, num_chunks);
			cudaReducePoly(a, num_chunks, irred_poly, indx_mask_bit);
		}
	}
}

__global__ void cudaCreateChunkProdArrayKernel( CUDA_BIGNUM *a, CUDA_BIGNUM *b, CUDA_BIGNUM *arr, uint32 numChunks )
{
	// thid_x is the column of the result array
	int32 thid_x = (blockIdx.x * blockDim.x) + threadIdx.x;
	// thid_y is the row of the result array
	int32 thid_y = (blockIdx.y * blockDim.y) + threadIdx.y;
	// thid inside of the current block
	// this is used to index the shared memory of the current thread
	int32 thid_inBlock = (threadIdx.y * blockDim.x) + threadIdx.x;

	// The temporary result of each thread:
	// -> shared memory is allocated for the entire block
	// -> 64 threads per block * 2 chunks per result
	// The temporary result is used to store the intermediate
	// result of the multiplication. If no shared memory
	// value would be used, every intermediate result has to
	// be stored in global memory which would result in 
	// decreased memory efficiency ans performance.
	__shared__ CUDA_BIGNUM res[2 * 64];
	// Because every block has to shift the value of a
	// it has to be stored in a temporary shared memory value.
	__shared__ CUDA_BIGNUM a_tmp[2 * 64];

	// Only exeute if the two dimensional thread indices
	// reference an element of the result array.
	if( thid_x < numChunks && thid_y < numChunks )
	{
		// Initialize the result with 0
		res[2 * thid_inBlock] 	  = 0;
		res[2 * thid_inBlock + 1] = 0;

		// Copy the value of a to a_tmp
		a_tmp[2 * thid_inBlock] 	= 0;
		a_tmp[2 * thid_inBlock + 1] = a[thid_x];

		// For every bit of the value b ...
		for( uint32 j=0; j<CUDA_BIGNUM_SIZE_BITS; ++j ) 
		{
			// ... check if the bit is set
			if( isbitset(b[thid_y], j) ) 
			{
				// If the bit in b is set -> add the current content of 
				// a_tmp to the temporary result
				res[2 * thid_inBlock] 	  ^= a_tmp[2 * thid_inBlock];
				res[2 * thid_inBlock + 1] ^= a_tmp[2 * thid_inBlock + 1];
			} 
			
			// Shift the content of a_tmp one bit to the left
			cudaBitShiftLeft1(&a_tmp[2 * thid_inBlock], 2);
		}

		// Copy the temporary result to the global result array
		arr[(thid_y + thid_x + 1) * 2 * numChunks + thid_x] = res[2 * thid_inBlock + 1];
		// Copy the carry to the global result array
		arr[(thid_y + thid_x) * 2 * numChunks + numChunks + thid_x] = res[2 * thid_inBlock];
	}
}

// TODO: Array Zeile des Blocks in Shared Memory puffern
__global__ void cudaParDiagBlockSumKernel( CUDA_BIGNUM *arr, uint32 numChunks ) 
{
	// thid_x is the column of the array
	int32 thid_x = (blockIdx.x * blockDim.x) + threadIdx.x;
	// thid_y is the row of the array
	int32 thid_y = (blockIdx.y * blockDim.y) + threadIdx.y;

	// Only exeute if the two dimensional thread indices
	// reference an element of the array.
	if( thid_x < numChunks && thid_y < 2 * numChunks )
	{
		// Each row consists of 2 * numChunks elements
		int32 chunksRow = 2 * numChunks;
		// rowOffset is the index of the first element
		// in row thid_y if the matrix arr is 
		// transformed to a 1D array
		int32 rowOffset = thid_y * chunksRow;
		// colOffset is the index of the column 
		// calculated over all blocks. Because every block
		// sums twice as much elements as the number of 
		// threads per block, the blockDix.x is multiplied 
		// by 2.
		int32 colOffset = blockIdx.x * blockDim.x * 2;
		// currArr is the part of the array used by the
		// current block in respect of the row and col offset
		// Example:
		// - Block 0: currArr = arr[0 ... 127]
		// - Block 1: currArr = arr[128 ... 255]
		// ...
		CUDA_BIGNUM *currArr = &arr[rowOffset + colOffset];
		// numElemLeftInRow is the number of elements left
		// in the current row, starting at the current
		// column offset.
		int32 numElemLeftInRow = chunksRow - colOffset;

		// there are 64 threads per block
		for( int32 offset = 1; offset <= blockDim.x; offset = offset<<1 )
		{
			if( threadIdx.x * 2 * offset + offset < numElemLeftInRow )	
		    	currArr[threadIdx.x * 2 * offset] ^= currArr[threadIdx.x * 2 * offset + offset];

		  	// only syncs the threads of a block
		  	// => kernel results in one result per block
		  	__syncthreads();
		}
	}
}

__global__ void cudaParChunkSumKernel( CUDA_BIGNUM *arr, uint32 numChunks, 
	uint32 numElements, uint32 offset, CUDA_BIGNUM *res ) 
{
	// thid is the row index of the array
	int32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	// Only exeute if the row indes reference 
	// a valid row of the array.
	if( thid < 2 * numChunks )
	{
		// currArr points at the first element
		// of the current row indexed by the thid
		CUDA_BIGNUM *currArr = &arr[thid * 2 * numChunks];
		// res is initialized with the first row entry.
		// This ensures that res containts valid data and
		// the other chunks can be added
		res[thid] = currArr[0];

		// Now the results of all blocks created by the
		// function cudaParDiagBlockSumKernel are summed up.
		// numElements is equal to the number of blocks
		// in a row, whereas offset is the space between two 
		// blocks represented as number of chunks.
		for( uint32 i = 1; i < numElements; ++i )
			res[thid] ^= currArr[i * offset];
	}
}

__global__ void cudaShiftRightKernel( 
	CUDA_BIGNUM *a, uint32 numChunksA, 
	uint32 numBitsToShiftInBlock /*b*/, 
	uint32 blockOffset /*o*/, 
	CUDA_BIGNUM maskLeft /*l*/, 
	CUDA_BIGNUM maskRight /*r*/, 
	CUDA_BIGNUM *res, uint32 numChunksRes )
{
	// thid is the chunk index of the result
	uint32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Only exeute if the index references
	// a valid chunk of res.
	if(thid < numChunksRes) 
	{
		// Clear current result chunk
		res[thid] = 0;

		// First the low part of the result chunk is created.
		// Therefor the calculated source chunk of a has to be valid.
		if( blockOffset < numChunksA + thid + 1 )
			res[thid] = (*(a + numChunksRes + thid - blockOffset) & maskRight) >> numBitsToShiftInBlock;

		// As second step the high part of the result chunk is added.
		// Again the source chunk of a has to be valid.
		if( blockOffset < numChunksA + thid )
			res[thid] |= (*(a + numChunksRes + thid - (blockOffset + 1)) & maskLeft) << (CUDA_BIGNUM_SIZE_BITS - numBitsToShiftInBlock);
	}
}

__global__ void cudaShiftLeftKernel( 
	CUDA_BIGNUM *a, uint32 numChunksA, 
	uint32 numBitsToShiftInBlock /*b*/, 
	uint32 blockOffset /*o*/, 
	CUDA_BIGNUM maskLeft /*l*/, 
	CUDA_BIGNUM maskRight /*r*/, 
	CUDA_BIGNUM *res, uint32 numChunksRes )
{
	// thid is the chunk index of the result
	uint32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Only exeute if the index references
	// a valid chunk of res.
	if( thid < numChunksRes ) 
	{
		// Clear current result chunk
		res[thid] = 0;

		// First the high part of the result chunk is created.
		// Therefor the calculated source chunk of a has to be valid.
		if( thid + blockOffset >= numChunksRes - numChunksA && 
			thid + blockOffset - (numChunksRes - numChunksA) < numChunksA )
			res[thid] = (*(a + thid + blockOffset - (numChunksRes - numChunksA)) & maskLeft) << numBitsToShiftInBlock;

		// As second step the kiw part of the result chunk is added.
		// Again the source chunk of a has to be valid.
		if( thid + blockOffset + 1 >= numChunksA - numChunksA &&
			thid + blockOffset - (numChunksRes - numChunksA) + 1 < numChunksA )
			res[thid] |= (*(a + thid + blockOffset - (numChunksRes - numChunksA) + 1) & maskRight) >> (CUDA_BIGNUM_SIZE_BITS - numBitsToShiftInBlock);
	}
}

__global__ void cudaGetFirstNonEmptyChunkIndexKernel( CUDA_BIGNUM *x, uint32 numChunks, CUDA_BIGNUM *foundChunkIndex )
{
	// thid is the chunk index of the result
	uint32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid == 0 )
		*foundChunkIndex = ~0u;

	if( thid < numChunks )
	{
		if( x[thid] > 0 )
		{
#ifdef LINUXINTEL64
			atomicMin((unsigned long long *)foundChunkIndex, (unsigned long long)thid);
#else
			atomicMin((unsigned int *)foundChunkIndex, (unsigned int)thid);
#endif
		}
	}
}

__global__ void cudaCalcElemDegreeKernel( CUDA_BIGNUM *x, uint32 numChunks, CUDA_BIGNUM *foundChunkIndex, CUDA_BIGNUM *elemDegree )
{
	if( *foundChunkIndex == ~((CUDA_BIGNUM)0) )
		*elemDegree = 0;
	else
	{
		CUDA_BIGNUM index = 1;
		CUDA_BIGNUM h = ~(~((CUDA_BIGNUM)0) >> 1); 
		while( !(x[*foundChunkIndex] & h) ) 
		{
			h >>= 1;
			++index;
		}

		*elemDegree = (numChunks - 1 - (*foundChunkIndex)) * CUDA_BIGNUM_SIZE_BITS + (CUDA_BIGNUM_SIZE_BITS - index);
	}
}

__global__ void cudaPrintKernel( CUDA_BIGNUM *x, uint32 num_chunks )
{
	printf("x = %u\n", *x);
}

__global__ void cudaSwapBytes( CUDA_BIGNUM *x, uint32 num_chunks )
{
	uint32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < num_chunks )
	{
		x[thid] = swapBytes(x[thid]);
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Sets x to 0
//
////////////////////////////////////////////////////////////////////////////////
__global__ void cudaSet0Kernel( CUDA_BIGNUM *x, uint32 length ) 
{
	uint32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < length )
	{
		x[thid] = 0;
	}
}

__global__ void cudaSet1Kernel( CUDA_BIGNUM *x, uint32 length ) 
{
	uint32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( thid < length - 1 )
	{
		x[thid] = 0;
	}
	else if( thid == length - 1 )
	{
		x[thid] = 1;
	}
}

__global__ void cudaMaskFirstChunkKernel( CUDA_BIGNUM *x, CUDA_BIGNUM mask )
{
	x[0] = x[0] & mask;
}

__global__ void cudaHexDumpKernel( char *desc, void *addr, int len )
{
	cudaHexDump(desc, addr, len);
}

// ////////////////////////////////////////////////////////////////////////////////
// //
// //	Copies a to b
// //
// ////////////////////////////////////////////////////////////////////////////////
// __global__ void cudaCopyKernel( CUDA_BIGNUM *a, uint32 num_chunks_a, CUDA_BIGNUM *b, uint32 num_chunks_b ) 
// {
// 	uint32 thid = (blockIdx.x * blockDim.x) + threadIdx.x;

// 	if( thid < num_chunks_b ) {
// 		if( thid < num_chunks_a )
// 			b[thid] = a[thid];
// 		else
// 			b[thid] = 0;
// 	}
// }

// ////////////////////////////////////////////////////////////////////////////////
// /*
// 	Device Functions
// */
// ////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
//
//	Shift a 1 Bit to the left
//
////////////////////////////////////////////////////////////////////////////////
__device__ void cudaBitShiftLeft1( CUDA_BIGNUM *a, uint32 num_chunks ) 
{
	CUDA_BIGNUM tmp = 0;
	CUDA_BIGNUM carry = 0;
	CUDA_BIGNUM carry_last = 0;

	CUDA_BIGNUM lmask = pow((double)2, (double)CUDA_BIGNUM_SIZE_BITS) - 1;
	CUDA_BIGNUM umask = pow((double)2, (double)CUDA_BIGNUM_SIZE_BITS - 1);

	for( uint32 i = num_chunks; i > 0; --i ) 
	{
		tmp = 0;
		tmp = a[i-1];
		carry_last = carry;
		carry = (tmp&umask) >> (CUDA_BIGNUM_SIZE_BITS - 1);
		tmp <<= 1;
		a[i-1] = (tmp&lmask) | carry_last;
	}
}

////////////////////////////////////////////////////////////////////////////////
//
//	Checks if the bit at pos bitnum is set (idx from right to left)
//
////////////////////////////////////////////////////////////////////////////////
__device__ bool isbitset( CUDA_BIGNUM val, uint32 bitnum ) 
{
	return (val & ((CUDA_BIGNUM)1 << bitnum)) != 0;
}

////////////////////////////////////////////////////////////////////////////////
//
//	Reduce the extended field polynomial
//	TODO: Algorithmus der nur die Chunks reduziert in denen irred_poly mind
//		  ein Bit == 1 besitzt
//
////////////////////////////////////////////////////////////////////////////////
__device__ void cudaReducePoly(
	CUDA_BIGNUM *value,
	uint32 num_chunks,
	CUDA_BIGNUM *irred_poly,
	uint32 indx_mask_bit
	) 
{
	if( isbitset(value[0], CUDA_BIGNUM_SIZE_BITS - indx_mask_bit) ) 
	{
		for( uint32 i=0; i<num_chunks; ++i )
		{
			value[i] ^= irred_poly[i];
		}
	}
}

__device__ void cudaHexDump( char *desc, void *addr, int len ) 
{
	int i;
	unsigned char buff[17];       // stores the ASCII data
	unsigned char *pc = (unsigned char *)addr;     // cast to make the code cleaner.

	// Output description if given.
	if (desc != NULL)
		printf ("%s:\n", desc);

	// Process every byte in the data.

	for (i = 0; i < len; i++) {
		// Multiple of 16 means new line (with line offset).

		if ((i % 16) == 0) {
		// Just don't print ASCII for the zeroth line.

			if (i != 0)
				printf ("  %s\n", buff);

			// Output the offset.

			printf ("  %04x ", i);
		}

		// Now the hex code for the specific character.

		printf (" %02x", pc[i]);

		// And store a printable ASCII character for later.

		if ((pc[i] < 0x20) || (pc[i] > 0x7e))
			buff[i % 16] = '.';
		else
			buff[i % 16] = pc[i];
    	
    	buff[(i % 16) + 1] = '\0';
	}

	// Pad out last line if not exactly 16 characters.

	while ((i % 16) != 0) {
		printf ("   ");
		i++;
	}

	// And print the final ASCII bit.

	printf ("  %s\n", buff);
}


__device__ uint16 swapBytes( uint16 val ) 
{
    return (val << 8) | (val >> 8 );
}

__device__ int16 swapBytes( int16 val ) 
{
    return (val << 8) | ((val >> 8) & 0xFF);
}

__device__ uint32 swapBytes( uint32 val )
{
    val = ((val << 8) & 0xFF00FF00 ) | ((val >> 8) & 0xFF00FF ); 
    return (val << 16) | (val >> 16);
}

__device__ int32 swapBytes( int32 val )
{
    val = ((val << 8) & 0xFF00FF00) | ((val >> 8) & 0xFF00FF ); 
    return (val << 16) | ((val >> 16) & 0xFFFF);
}

__device__ int64 swapBytes( int64 val )
{
    val = ((val << 8) & 0xFF00FF00FF00FF00ULL ) | ((val >> 8) & 0x00FF00FF00FF00FFULL );
    val = ((val << 16) & 0xFFFF0000FFFF0000ULL ) | ((val >> 16) & 0x0000FFFF0000FFFFULL );
    return (val << 32) | ((val >> 32) & 0xFFFFFFFFULL);
}

__device__ uint64 swapBytes( uint64 val )
{
    val = ((val << 8) & 0xFF00FF00FF00FF00ULL ) | ((val >> 8) & 0x00FF00FF00FF00FFULL );
    val = ((val << 16) & 0xFFFF0000FFFF0000ULL ) | ((val >> 16) & 0x0000FFFF0000FFFFULL );
    return (val << 32) | (val >> 32);
}